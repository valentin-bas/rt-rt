#include "hip/hip_runtime.h"
#include "RaytracerTypes.cuh"

__device__ void vec_sub (VECTOR3D *v1, VECTOR3D *v2, VECTOR3D *v3) {

	v1->x = v2->x - v3->x;
	v1->y = v2->y - v3->y;
	v1->z = v2->z - v3->z;
}

__device__ void vec_add (VECTOR3D *v1, VECTOR3D *v2, VECTOR3D *v3) {
	
	v1->x = v2->x + v3->x;
	v1->y = v2->y + v3->y;
	v1->z = v2->z + v3->z;
}

__device__ void vec_scale (double scale, VECTOR3D *v1, VECTOR3D *v2) {
	
	v1->x = scale * v2->x;
	v1->y = scale * v2->y;
	v1->z = scale * v2->z;
}

__device__ double dotproduct (VECTOR3D *v1, VECTOR3D *v2) {
	
	return v1->x * v2->x + v1->y * v2->y + v1->z * v2->z;
}

__device__ VECTOR3D crossProduct(VECTOR3D *v1, VECTOR3D *v2) {
	VECTOR3D temp;
	temp.x =   ( (v1->y * v2->z) - (v1->z * v2->y) );
	temp.y = - ( (v1->x * v2->z) - (v1->z * v2->x) );
	temp.z =   ( (v1->x * v2->y) - (v1->y * v2->x) );
	return temp;
}


__device__ void normalize_vector (VECTOR3D *v) {
	
	double magnitude;
	
	// 1. calculate the magnitude (lerngth):
	magnitude = sqrt( dotproduct(v, v) );
	
	// 2. normalize the vector:
	v->x = v->x / magnitude;
	v->y = v->y / magnitude;
	v->z = v->z / magnitude;
}

__device__ void set_rgb_array(double* rgb_array, double cred, double cgreen, double cblue)
{
	rgb_array[CRED] = cred;
	rgb_array[CGREEN] = cgreen;
	rgb_array[CBLUE] = cblue;
}