#include "hip/hip_runtime.h"
#include "RaytracerTypes.cuh"
#include "RaytracerDevice.cuh"

__global__ void init_kernel(int width, int height)
{
	viewport.xvmin = -width / 2;
	viewport.yvmin = -height / 2;
	viewport.xvmax = width / 2;
	viewport.yvmax = height / 2;
	camera_frame.u.x = 1.0;
	camera_frame.u.y = 0.0;
	camera_frame.u.z = 0.0;
	camera_frame.v.x = 0.0;
	camera_frame.v.y = 1.0;
	camera_frame.v.z = 0.0;
	camera_frame.n.x = 0.0;
	camera_frame.n.y = 0.0;
	camera_frame.n.z = 1.0;
	view_point.x = (viewport.xvmax - viewport.xvmin) / 2.0 ;
	view_point.y = (viewport.yvmax - viewport.yvmin) / 2.0 ;
	view_point.z = 0.0;
	static_view_point=view_point;
	focal_distance = FOCALDIST;
	//SCENE
	light.x = view_point.x - 1300;
	light.y = view_point.y + 1300;
	light.z = view_point.z - 300;
	sphere[0].radius = RADIUS/1.5;
	sphere[0].center.x  = view_point.x - (RADIUS+30);
	sphere[0].center.y  = view_point.y ;
	sphere[0].center.z  = view_point.z - focal_distance - (2*RADIUS+20);
	set_rgb_array(sphere[0].kd_rgb, 0.0, 0.0, 0.8);
	set_rgb_array(sphere[0].ks_rgb, 1.0, 1.0, 1.0);
	set_rgb_array(sphere[0].ka_rgb, 0.0, 0.0, 0.2);
	set_rgb_array(sphere[0].kr_rgb, 0.0, 0.0, 0.0);
	sphere[0].shininess = 100.0;
	sphere[0].refraction_index=1.52;
	sphere[0].mirror = false;
	sphere[1].radius = RADIUS/1.2;
	sphere[1].center.x  = view_point.x + 0;
	sphere[1].center.y  = view_point.y + 50;
	sphere[1].center.z  = view_point.z - focal_distance - (3*RADIUS+20);
	set_rgb_array(sphere[1].kd_rgb, 0.0, 0.8, 0.0);
	set_rgb_array(sphere[1].ks_rgb, 0.5, 0.5, 0.5);
	set_rgb_array(sphere[1].ka_rgb, 0.0, 0.2, 0.0);
	set_rgb_array(sphere[1].kr_rgb, 0.5, 0.5, 0.5);
	sphere[1].shininess = 10.0;
	sphere[1].refraction_index=1.52;
	sphere[1].mirror = false;
	sphere[2].radius = RADIUS;
	sphere[2].center.x  = view_point.x + (2*RADIUS+30);
	sphere[2].center.y  = view_point.y + 100;
	sphere[2].center.z  = view_point.z - focal_distance - (4*RADIUS+20);
	set_rgb_array(sphere[2].kd_rgb, 0.8, 0.0, 0.0);
	set_rgb_array(sphere[2].ks_rgb, 0.7, 0.7, 0.7);
	set_rgb_array(sphere[2].ka_rgb, 0.2, 0.0, 0.0);
	set_rgb_array(sphere[2].kr_rgb, 0.3, 0.3, 0.3);
	sphere[2].shininess = 100.0;
	sphere[2].refraction_index=1.52;
	sphere[2].mirror = false;
	sphere[3].radius = 100*RADIUS;
	sphere[3].center.x  = view_point.x ;
	sphere[3].center.y  = view_point.y - 100*RADIUS-130;
	sphere[3].center.z  = view_point.z - focal_distance - (4*RADIUS+20);
	set_rgb_array(sphere[3].kd_rgb, 0.2, 0.2, 0.2);
	set_rgb_array(sphere[3].ks_rgb, 0.8, 0.8, 0.5);
	set_rgb_array(sphere[3].ka_rgb, 0.0, 0.0, 0.0);
	set_rgb_array(sphere[3].kr_rgb, 0.2, 0.2, 0.5);
	sphere[3].shininess = 100.0;
	sphere[3].refraction_index=1.52;
	sphere[3].mirror = true;
};

__global__ void raytracer_kernel(float3* pos, unsigned int width, unsigned int height)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i>= width || j>= height)
		return;

	RAY ray, shadow_ray;
	PIXEL pixel;
	int hitIdx = -1;
	double current_lambda = 0x7fefffffffffffff;
	SPHERE_INTERSECTION intersection, current_intersection;

	pixel.i = i;
	pixel.j = j;
	compute_ray(&ray, &view_point, &viewport, &pixel, &camera_frame, focal_distance);
	for (int k=0; k < NSPHERES; k++)
	{
		if (sphere_intersection(&ray, &sphere[k], &intersection))
		{
			intersection_normal(&sphere[k], &intersection, &ray);
			if (intersection.lambda_in < current_lambda)
			{
				current_lambda=intersection.lambda_in;
				hitIdx=k;
				current_intersection=intersection;
			}
		}
	}

	SPHERE_INTERSECTION shadow_ray_intersection;
	double theta;
	double red, green, blue;
	bool shadow = false;
	Color color;

	if (hitIdx > -1)
	{
		compute_shadow_ray(&shadow_ray, &current_intersection , &light);
		theta = dotproduct(&(shadow_ray.direction), &(current_intersection.normal));
		for (int l=0; l < NSPHERES; l++)
		{
			if (l != hitIdx)
			{
				if (sphere_intersection(&shadow_ray, &sphere[l], &shadow_ray_intersection) && (theta > 0.0))
					shadow=true;
			}
		}
		red=green=blue=0;
		if (shadow)
		{
			red += shadowColor(sphere[hitIdx].ka_rgb[CRED], 1.0);
			green += shadowColor(sphere[hitIdx].ka_rgb[CGREEN], 1.0);
			blue += shadowColor(sphere[hitIdx].ka_rgb[CBLUE], 1.0);
		}
		else
		{
			SPHERE* s = &sphere[hitIdx];
			red += blinnphongColor(&current_intersection, &light, &view_point,
								   s->kd_rgb[CRED], s->ks_rgb[CRED], s->ka_rgb[CRED],
								   s->shininess, 1.0, 1.0);
			green += blinnphongColor(&current_intersection, &light, &view_point,
								   s->kd_rgb[CGREEN], s->ks_rgb[CGREEN], s->ka_rgb[CGREEN],
								   s->shininess, 1.0, 1.0);
			blue += blinnphongColor(&current_intersection, &light, &view_point,
								   s->kd_rgb[CBLUE], s->ks_rgb[CBLUE], s->ka_rgb[CBLUE],
								   s->shininess, 1.0, 1.0);
		}
		if (red>1.0)
			red=1.0;
		if (green>1.0)
			green=1.0;
		if (blue>1.0)
			blue=1.0;
		//double* colorTmp = sphere[intersection_object].kd_rgb;
		//color.components = make_uchar4(colorTmp[0] * 255, colorTmp[1] * 255, colorTmp[2] * 255, 1);
		color.components = make_uchar4(red * 255, green * 255, blue * 255, 1);
	}
	else
		color.components = make_uchar4(0, 0, 0, 1);
	pos[j * width + i] = make_float3(i, j, color.c);
}

__device__ float timer=0.0f;

__global__ void animate_kernel()
{
	sphere[0].center.y=view_point.y+sinf(timer)*100;
	sphere[1].center.y=view_point.y+50+sinf(2*timer)*100;
	sphere[2].center.y=view_point.y+100+sinf(1.5*timer)*100;
	timer+=0.02f;
}