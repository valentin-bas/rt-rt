#include "hip/hip_runtime.h"
#include "Raytracer.h"

#include "RaytracerTypes.cuh"
#include "RaytracerGlobal.cuh"

#include <cuda_gl_interop.h>


Raytracer::Raytracer(GLRenderer* renderer, unsigned int priority) :
	DisplayableTask(priority), _buffer(nullptr), _vbo(GL_INVALID_VALUE)
{
	_renderer = renderer;
}

Raytracer::~Raytracer()
{
}

bool
Raytracer::Start()
{
	unsigned int width = _renderer->Width();
	unsigned int height = _renderer->Height();
	init_kernel<<<1,1>>>(width, height);
	if (hipMalloc(&_buffer, width * height * sizeof(uchar4)) != hipSuccess)
		return false;
	glGenBuffers(1, &_vbo);
	if (_vbo == GL_INVALID_VALUE)
		return false;
	glBindBuffer(GL_ARRAY_BUFFER, _vbo);
	unsigned int size = width * height * sizeof(float3);
	glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	cudaGLRegisterBufferObject(_vbo);
	return true;
}

void
Raytracer::Update()
{
	unsigned int width = _renderer->Width();
	unsigned int height = _renderer->Height();

	animate_kernel<<<1,1>>>();
	hipDeviceSynchronize();
	float3 *dptr;
    cudaGLMapBufferObject((void**)&dptr, _vbo);
	dim3 block(32, 16, 1);
	dim3 grid(width / block.x + 1, height / block.y + 1, 1);
	hipDeviceSynchronize();
	raytracer_kernel<<<grid,block>>>(dptr, width, height);
	cudaGLUnmapBufferObject(_vbo);
}

void
Raytracer::Display()
{
	unsigned int width = _renderer->Width();
	unsigned int height = _renderer->Height();

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(0.0f, width, 0.0f, height, -1.0f, 1.0f);

	glBindBuffer(GL_ARRAY_BUFFER, _vbo);
    glVertexPointer(2, GL_FLOAT, 12, 0);
	glColorPointer(4, GL_UNSIGNED_BYTE, 12, (GLvoid*)8);
    glEnableClientState(GL_VERTEX_ARRAY);
	glEnableClientState(GL_COLOR_ARRAY);
	glDrawArrays(GL_POINTS, 0, width * height);
    glDisableClientState(GL_VERTEX_ARRAY);
}

void
Raytracer::Stop()
{
	if (_buffer != nullptr)
		hipFree(_buffer);
	if (_vbo != GL_INVALID_VALUE)
		glDeleteBuffers(1, &_vbo);
}