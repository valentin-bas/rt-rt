#include "hip/hip_runtime.h"
#include "RaytracerTypes.cuh"
#include "RaytracerDevice.cuh"

__device__ double focal_distance;
__device__ VEC_BASIS camera_frame;
__device__ VECTOR3D view_point, static_view_point;
__device__ VIEWPORT viewport;
__device__ SPHERE sphere[NSPHERES];
__device__ VECTOR3D light;

__device__ void compute_ray(RAY* ray, VECTOR3D* view_point, VIEWPORT* viewport, PIXEL* pixel, VEC_BASIS* camera_frame, double distance)
{
	float u, v;
	VECTOR3D v1, v2, v3, v4, dir;
	
	
	// 1. calculate u and v coordinates of the pixels on the image plane:
	u = (double)(viewport->xvmin) + (double)(pixel->i) + 0.5 ;  
	v = (double)(viewport->yvmin) + (double)(pixel->j) + 0.5 ;  
	
	// 2. calculate ray direction
	vec_scale(-distance, &v1, &camera_frame->n);
	vec_scale(u, &v2, &camera_frame->u);
	vec_scale(v, &v3, &camera_frame->v);
	
	ray->origin.x = view_point->x;  
	ray->origin.y = view_point->y;
	ray->origin.z = view_point->z;
	
	vec_add(&v4, &v1, &v2);
	vec_add(&dir, &v4, &v3);
	normalize_vector(&dir);
	
	ray->direction.x = dir.x;
	ray->direction.y = dir.y;
	ray->direction.z = dir.z;
}

__device__ bool sphere_intersection (RAY *ray, SPHERE *sphere, SPHERE_INTERSECTION* intersection)
{
	double discriminant;
	double A, B, C;
	double lambda1, lambda2;
	VECTOR3D temp;
	
	A = dotproduct(&ray->direction, &ray->direction);
	vec_sub(&temp, &ray->origin, &sphere->center);
	B = 2 * dotproduct(&temp, &ray->direction);
	vec_sub(&temp, &ray->origin, &sphere->center);
	C = dotproduct(&temp, &temp) - (sphere->radius * sphere->radius);
	discriminant = B*B - 4*A*C;
	if (discriminant >= 0)
	{
		lambda1 = (-B + sqrt(discriminant)) / (2*A);
		lambda2 = (-B - sqrt(discriminant)) / (2*A);
		// is the object visible from the eye (lambda1,2>0)
		if (lambda1>=0 && lambda2>=0)
		{
			if (lambda1 == lambda2)
				intersection->lambda_in = intersection->lambda_out = lambda1;
			else if (lambda1 < lambda2)
			{
				intersection->lambda_in  = lambda1;
				intersection->lambda_out = lambda2;
			}
			else
			{
				intersection->lambda_in  = lambda2;
				intersection->lambda_out = lambda1;
			}
			intersection->valid = true;
			return true;
		}
		else
		{
			intersection->valid = false;
			return false;
		}
	}
	else
	{
		intersection->valid = false;
		return false;
	}
}

__device__ void intersection_normal(SPHERE *sphere, SPHERE_INTERSECTION* intersection, RAY* ray)
{
	double lambda, scale;
	VECTOR3D v1, v2, point, normal;
	
	lambda = intersection->lambda_in;
	vec_scale(lambda, &v1, &ray->direction);
	vec_add(&point, &v1, &ray->origin);
	intersection->point.x = point.x;
	intersection->point.y = point.y;
	intersection->point.z = point.z;
	vec_sub(&v2, &point, &sphere->center);
	scale = 1.0 / sphere->radius;
	vec_scale(scale, &normal, &v2);
	normalize_vector(&normal);
	intersection->normal.x = normal.x;
	intersection->normal.y = normal.y;
	intersection->normal.z = normal.z;
}

__device__ void compute_shadow_ray(RAY* ray, SPHERE_INTERSECTION* intersection, VECTOR3D* light)
{
	VECTOR3D dir;
	
	// ray origin is in the intersection point
	ray->origin.x = intersection->point.x;
	ray->origin.y = intersection->point.y;
	ray->origin.z = intersection->point.z;
	// ray direction is from the intersection point towards the light:
	vec_sub(&dir, light, &intersection->point);
	normalize_vector(&dir);
	ray->direction.x = dir.x;
	ray->direction.y = dir.y;
	ray->direction.z = dir.z;
}

__device__ double shadowColor(double ka, double amb_intensity)
{
	
	return (ka * amb_intensity);	
}

__device__ double blinnphongColor(SPHERE_INTERSECTION *intersection, VECTOR3D* light, VECTOR3D* viewpoint,
								  double kd, double ks, double ka, double p, double intensity, double amb_intensity)
{
	
	double color_diffuse = 0.0; 
	double color_specular = 0.0;
	VECTOR3D l;
	VECTOR3D h;
	VECTOR3D v;

	// compute vector v :
	vec_sub(&v, viewpoint, &intersection->point);
	normalize_vector(&v);
	// compute vector l :
	vec_sub(&l, light, &intersection->point);
	normalize_vector(&l);
	// compute vector h:
	vec_add(&h, &v, &l);
	normalize_vector(&h);
	// compute the diffuse intensity:
	color_diffuse = kd * intensity * dotproduct(&l, &intersection->normal) ;
	if (color_diffuse < 0.0)
		color_diffuse = 0.0;
	// compute the specular intensity:
	color_specular = ks * intensity * pow (dotproduct(&h, &intersection->normal), p);
	if (color_specular < 0.0)
		color_specular = 0.0;
	return (color_diffuse + color_specular + (ka * amb_intensity));	
}